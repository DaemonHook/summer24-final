#include "CudaCheckError.h"
#include "CudaGraph.h"
#include <iostream>
#include <algorithm>

CudaLinkGraph::CudaLinkGraph(LinkGraph& memoryGraph)
{
    nodeNum = memoryGraph.getNodeNum();
    edgeNum = memoryGraph.getEdgeNum();

    // startEdgeIndices相当于原始va数组
    std::vector<nodeId_t> h_startEdgeIndices = memoryGraph.va;
    std::vector<nodeId_t> h_endEdgeIndices(nodeNum);

    for (nodeId_t i = 0; i < nodeNum; i++) {
        // 没有边的特殊处理
        if (h_startEdgeIndices[i] == NO_EDGE) {
            h_endEdgeIndices[i] = NO_EDGE;
            continue;
        }
        // 逐个去找终点
        nodeId_t curEdgeIndex = h_startEdgeIndices[i];
        nodeId_t nextNodeIndex = i + 1;
        while (nextNodeIndex < nodeNum && h_startEdgeIndices[nextNodeIndex] == NO_EDGE) {
            nextNodeIndex++;
        }
        // 如果超过了最后一个节点
        if (nextNodeIndex == nodeNum) {
            h_endEdgeIndices[i] = edgeNum;
        } else {
            h_endEdgeIndices[i] = h_startEdgeIndices[nextNodeIndex];
        }
    }
    
    checkError(hipMalloc(&d_edgeIndicesStart, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_edgeIndicesStart, h_startEdgeIndices.data(), nodeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_edgeIndicesEnd, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_edgeIndicesEnd, h_endEdgeIndices.data(), nodeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_ea, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_ea, memoryGraph.ea.data(), edgeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_weights, edgeNum * sizeof(weight_t)));
    checkError(hipMemcpy(d_weights, memoryGraph.weights.data(), edgeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
}

CudaLinkGraph::~CudaLinkGraph()
{
    checkError(hipFree(d_edgeIndicesStart));
    checkError(hipFree(d_edgeIndicesEnd));
    checkError(hipFree(d_ea));
    checkError(hipFree(d_weights));
}

CudaMatGraph::CudaMatGraph(MatrixGraph& graph)
{
    nodeNum = graph.getNodeNum();
    checkError(hipMalloc(&d_mat, nodeNum * nodeNum * sizeof(weight_t)));
    checkError(hipMemcpy(d_mat, graph._mat.data(), nodeNum * nodeNum * sizeof(weight_t), hipMemcpyHostToDevice));
}

CudaMatGraph::~CudaMatGraph()
{
    checkError(hipFree(d_mat));
}
