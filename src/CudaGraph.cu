#include "CudaCheckError.h"
#include "CudaGraph.h"

__host__ void convertLinkGraph(LinkGraph& linkGraph, nodeId_t& nodeNum, long& edgeNum,
    nodeId_t*& startIndices, nodeId_t*& endIndices, nodeId_t*& ea, weight_t*& weights)
{
    nodeNum = linkGraph.getNodeNum();
    edgeNum = linkGraph.getEdgeNum();
    startIndices = new nodeId_t[nodeNum];
    endIndices = new nodeId_t[nodeNum];
    ea = new nodeId_t[edgeNum];
    weights = new weight_t[edgeNum];
    memcpy(startIndices, linkGraph.va.data(), linkGraph.va.size() * sizeof(nodeId_t));

    for (nodeId_t i = 0; i < linkGraph.getNodeNum(); i++) {
        nodeId_t curIndex = startIndices[i];
        if (i == linkGraph.getNodeNum() - 1) {
        }
    }
}

CudaLinkGraph::CudaLinkGraph(LinkGraph& memoryGraph)
{
    nodeNum = memoryGraph.getNodeNum();
    edgeNum = memoryGraph.getEdgeNum();

    // startEdgeIndices相当于原始va数组
    std::vector<nodeId_t> h_startEdgeIndices = memoryGraph.va;

    std::vector<nodeId_t> h_endEdgeIndices(nodeNum);
    for (nodeId_t i = 0; i < nodeNum; i++) {
        // 没有边的特殊处理
        if (h_startEdgeIndices[i] == NO_EDGE) {
            h_endEdgeIndices[i] = NO_EDGE;
            continue;
        }
        // 逐个去找终点
        nodeId_t curEdgeIndex = h_startEdgeIndices[i];
        nodeId_t nextNodeIndex = i + 1;
        while (nextNodeIndex < nodeNum && h_startEdgeIndices[i] != NO_EDGE) {
            nextNodeIndex++;
        }
        // 如果超过了最后一个节点
        if (nextNodeIndex == nodeNum) {
            h_endEdgeIndices[i] = edgeNum;
        } else {
            h_endEdgeIndices[i] = h_startEdgeIndices[nextNodeIndex];
        }
    }

    checkError(hipMalloc(&d_edgeIndicesStart, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_edgeIndicesStart, h_startEdgeIndices.data(), nodeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_edgeIndicesEnd, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_edgeIndicesEnd, h_endEdgeIndices.data(), nodeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_ea, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_ea, memoryGraph.ea.data(), edgeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_weights, edgeNum * sizeof(weight_t)));
    checkError(hipMemcpy(d_weights, memoryGraph.weights.data(), edgeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
}

CudaLinkGraph::~CudaLinkGraph()
{
    checkError(hipFree(d_edgeIndicesStart));
    checkError(hipFree(d_edgeIndicesEnd));
    checkError(hipFree(d_ea));
    checkError(hipFree(d_weights));
}
